#include <cstdlib>
#include <iostream>
#include <hip/hip_runtime.h>

static inline void check(hipError_t err, const char *context)
{
  if (err != hipSuccess)
  {
    std::cerr << "CUDA error: " << context << ": "
              << hipGetErrorString(err) << std::endl;
    std::exit(EXIT_FAILURE);
  }
}

#define CHECK(x) check(x, #x)

static inline int divup(int a, int b)
{
  return (a + b - 1) / b;
}

inline int static roundup(int a, int b)
{
  return divup(a, b) * b;
}

__global__ void precompute(int ny, int nx, const float *data, float *diffs)
{
  int y = blockIdx.x * blockDim.x + threadIdx.x;
  if (y >= ny)
  {
    return;
  }

  float mean = 0.0f;
  for (int x = 0; x < nx; ++x)
  {
    mean += data[x + y * nx];
  }
  mean /= nx;

  float sum = 0.0f;
  for (int x = 0; x < nx; ++x)
  {
    float diff = data[x + y * nx] - mean;
    diffs[y + x * ny] = diff;
    sum += diff * diff;
  }

  for (int x = 0; x < nx; ++x)
  {
    diffs[y + x * ny] /= sqrt(sum);
  }
}

__global__ void compute(int ny, int nx, float *diffs, float *result)
{
  int ia = threadIdx.x;
  int ja = threadIdx.y;
  int ic = blockIdx.x;
  int jc = blockIdx.y;

  float sums[8][8] = {0};

  for (int k = 0; k < nx; ++k)
  {
    float x[8];
    float y[8];
    for (int ib = 0; ib < 8; ++ib)
    {
      int i = ic * 64 + ib * 8 + ia;
      x[ib] = diffs[ny * k + i];
      for (int jb = 0; jb < 8; ++jb)
      {
        int j = jc * 64 + jb * 8 + ja;
        y[jb] = diffs[ny * k + j];
      }
    }

    for (int ib = 0; ib < 8; ++ib)
    {
      for (int jb = 0; jb < 8; ++jb)
      {
        sums[ib][jb] += x[ib] * y[jb];
      }
    }
  }

  for (int ib = 0; ib < 8; ++ib)
  {
    for (int jb = 0; jb < 8; ++jb)
    {
      int i = ic * 64 + ib * 8 + ia;
      int j = jc * 64 + jb * 8 + ja;
      if (i < ny && j < ny)
      {
        result[i + j * ny] = sums[ib][jb];
      }
    }
  }
}

/*
This is the function you need to implement. Quick reference:
- input rows: 0 <= y < ny
- input columns: 0 <= x < nx
- element at row y and column x is stored in data[x + y*nx]
- correlation between rows i and row j has to be stored in result[i + j*ny]
- only parts with 0 <= j <= i < ny need to be filled
*/
void correlate(int ny, int nx, const float *data, float *result)
{
  float *d_data, *d_result, *d_diffs;

  constexpr int block_size = 8;
  int rows_after_padding = roundup(ny, block_size * block_size);

  size_t data_size = ny * nx * sizeof(float);
  size_t result_size = ny * ny * sizeof(float);
  size_t diffs_size = rows_after_padding * nx * sizeof(float);

  CHECK(hipMalloc((void **)&d_data, data_size));
  CHECK(hipMalloc((void **)&d_result, result_size));
  CHECK(hipMalloc((void **)&d_diffs, diffs_size));

  CHECK(hipMemset(d_data, 0, data_size));
  CHECK(hipMemset(d_result, 0, result_size));
  CHECK(hipMemset(d_diffs, 0, diffs_size));

  CHECK(hipMemcpy(d_data, data, data_size, hipMemcpyHostToDevice));

  {
    dim3 dim_block(64);
    dim3 dim_grid(divup(ny, dim_block.x));
    precompute<<<dim_grid, dim_block>>>(ny, nx, d_data, d_diffs);
    CHECK(hipGetLastError());
  }

  {
    dim3 dim_block(block_size, block_size);
    dim3 dim_grid(rows_after_padding / 64, rows_after_padding / 64);
    compute<<<dim_grid, dim_block>>>(ny, nx, d_diffs, d_result);
    CHECK(hipGetLastError());
  }

  CHECK(hipMemcpy(result, d_result, result_size, hipMemcpyDeviceToHost));

  CHECK(hipFree(d_data));
  CHECK(hipFree(d_result));
  CHECK(hipFree(d_diffs));
}