#include <cstdlib>
#include <iostream>
#include <hip/hip_runtime.h>

static inline void check(hipError_t err, const char *context)
{
  if (err != hipSuccess)
  {
    std::cerr << "CUDA error: " << context << ": "
              << hipGetErrorString(err) << std::endl;
    std::exit(EXIT_FAILURE);
  }
}

#define CHECK(x) check(x, #x)

static inline int divup(int a, int b)
{
  return (a + b - 1) / b;
}

__global__ void precompute(int ny, int nx, const float *data, float *diffs)
{
  int y = blockIdx.x * blockDim.x + threadIdx.x;
  if (y >= ny)
  {
    return;
  }

  float mean = 0.0f;
  for (int x = 0; x < nx; ++x)
  {
    mean += data[x + y * nx];
  }
  mean /= nx;

  float sum = 0.0f;
  for (int x = 0; x < nx; ++x)
  {
    float diff = data[x + y * nx] - mean;
    diffs[x + y * nx] = diff;
    sum += diff * diff;
  }

  for (int x = 0; x < nx; ++x)
  {
    diffs[x + y * nx] /= sqrt(sum);
  }
}

__global__ void compute(int ny, int nx, float *diffs, float *result)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;

  if (i >= ny || j > i)
  {
    return;
  }

  float sum[8][8] = {0};

  for (int x = 0; x < nx; ++x)
  {
    for (int ii = 0; ii < 8; ++ii)
    {
      for (int jj = 0; jj < 8; ++jj)
      {
        int iii = i * 8 + ii;
        int jjj = j * 8 + jj;
        if (iii >= ny || jjj > iii)
        {
          continue;
        }
        sum[ii][jj] += diffs[x + iii * nx] * diffs[x + jjj * nx];
      }
    }
  }

  for (int ii = 0; ii < 8; ++ii)
  {
    for (int jj = 0; jj < 8; ++jj)
    {
      int iii = i * 8 + ii;
      int jjj = j * 8 + jj;
      if (iii >= ny || jjj > iii)
      {
        continue;
      }
      result[iii + jjj * ny] = sum[ii][jj];
    }
  }
}

/*
This is the function you need to implement. Quick reference:
- input rows: 0 <= y < ny
- input columns: 0 <= x < nx
- element at row y and column x is stored in data[x + y*nx]
- correlation between rows i and row j has to be stored in result[i + j*ny]
- only parts with 0 <= j <= i < ny need to be filled
*/
void correlate(int ny, int nx, const float *data, float *result)
{
  float *d_data, *d_result, *d_diffs;

  constexpr int block_size = 8;
  int blocks_of_rows = divup(ny, block_size);
  int rows_after_padding = blocks_of_rows * block_size;

  size_t data_size = ny * nx * sizeof(float);
  size_t result_size = ny * ny * sizeof(float);
  size_t diffs_size = rows_after_padding * nx * sizeof(float);

  CHECK(hipMalloc((void **)&d_data, data_size));
  CHECK(hipMalloc((void **)&d_result, result_size));
  CHECK(hipMalloc((void **)&d_diffs, diffs_size));

  CHECK(hipMemset(d_data, 0, data_size));
  CHECK(hipMemset(d_result, 0, result_size));
  CHECK(hipMemset(d_diffs, 0, diffs_size));

  CHECK(hipMemcpy(d_data, data, data_size, hipMemcpyHostToDevice));

  {
    dim3 dim_block(32);
    dim3 dim_grid(divup(ny, dim_block.x));
    precompute<<<dim_grid, dim_block>>>(ny, nx, d_data, d_diffs);
    CHECK(hipGetLastError());
  }

  {
    dim3 dim_block(block_size, block_size);
    dim3 dim_grid(divup(blocks_of_rows, dim_block.x), divup(blocks_of_rows, dim_block.y));
    compute<<<dim_grid, dim_block>>>(ny, nx, d_diffs, d_result);
    CHECK(hipGetLastError());
  }

  CHECK(hipMemcpy(result, d_result, result_size, hipMemcpyDeviceToHost));

  CHECK(hipFree(d_data));
  CHECK(hipFree(d_result));
  CHECK(hipFree(d_diffs));
}